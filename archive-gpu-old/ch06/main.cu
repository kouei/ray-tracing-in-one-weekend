#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hittable_list.h"
#include "camera.h"
#include "cuda_utility.h"

constexpr const float TMAX = std::numeric_limits<float>::max();

__device__ vec3 ray_color(const ray & r, hittable_ptr_t world) {
    hit_record rec;
    if (world->hit(r, 0.0f, TMAX, rec)) {
        return 0.5f * (rec.normal + vec3(1.0f, 1.0f, 1.0f));
    }
    else {
        vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5f * (unit_direction.y() + 1.0f);
        return (1.0f - t) * vec3(1.0f, 1.0f, 1.0f) + t * vec3(0.5f, 0.7f, 1.0f);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState * rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) {
        return;
    }

    int pixel_index = j * max_x + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 * frame_buffer,
                       int max_x, int max_y, int ns,
                       camera_ptr_t * cam, hittable_ptr_t * world, hiprandState * rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) {
        return;
    }

    int pixel_index = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 pixel_color(0.0f, 0.0f, 0.0f);
    for(int s = 0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = cam[0]->get_ray(u, v);
        pixel_color += ray_color(r, world[0]);
    }

    frame_buffer[pixel_index] = pixel_color / float(ns);
}

__global__ void create_world(hittable_ptr_t * d_list, hittable_ptr_t * d_world, camera_ptr_t * d_camera) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_list[0] = new sphere(vec3(0.0f, 0.0f, -1.0f), 0.5f);
        d_list[1] = new sphere(vec3(0.0f, -100.5f, -1.0f), 100.0f);
        d_world[0] = new hittable_list(d_list, 2);
        d_camera[0] = new camera();
    }
}

__global__ void free_world(hittable_ptr_t * d_list, hittable_ptr_t * d_world, camera_ptr_t * d_camera) {
    delete d_list[0];
    delete d_list[1];
    delete d_world[0];
    delete d_camera[0];
}

int main() {
    int nx = 1200;
    int ny = 600;
    int ns = 100;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny;
    vec3 * frame_buffer;
    size_t frame_buffer_size = num_pixels * sizeof(*frame_buffer);

    // Allocate Frame Buffer
    checkCudaErrors(hipMallocManaged(&frame_buffer, frame_buffer_size));

    // Allocate Random State
    hiprandState * d_rand_state;
    checkCudaErrors(hipMalloc(&d_rand_state, num_pixels * sizeof(*d_rand_state)));

    // make our world of hitables & the camera
    hittable_ptr_t * d_list;
    checkCudaErrors(hipMalloc(&d_list, 2 * sizeof(*d_list)));
    hittable_ptr_t * d_world;
    checkCudaErrors(hipMalloc(&d_world, sizeof(*d_world)));
    camera_ptr_t * d_camera;
    checkCudaErrors(hipMalloc(&d_camera, sizeof(*d_camera)));
    create_world<<<1, 1>>>(d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    auto start = std::chrono::high_resolution_clock::now();

    // Render our buffer
    int block_dim_x = (nx + tx - 1) / tx;
    int block_dim_y = (ny + ty - 1) / ty;
    dim3 blocks(block_dim_x, block_dim_y);
    dim3 threads(tx, ty);
    
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(frame_buffer, nx, ny,  ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    

    auto end = std::chrono::high_resolution_clock::now();

    auto timer_in_ms = std::chrono::duration<float, std::milli>(end - start);
    std::cerr << "Time cost: " << timer_in_ms.count() << "ms.\n";
    
    // Output FB as Image
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * frame_buffer[pixel_index].r());
            int ig = int(255.99 * frame_buffer[pixel_index].g());
            int ib = int(255.99 * frame_buffer[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1, 1>>>(d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(frame_buffer));

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();
}