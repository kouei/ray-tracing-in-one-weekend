#include "hip/hip_runtime.h"
#include "camera.h"
#include "color.h"
#include "cuda_utility.h"
#include "hittable.h"
#include "hittable_list.h"
#include "interval.h"
#include "ray.h"
#include "rtweekend.h"
#include "sphere.h"
#include "vec3.h"
#include <chrono>
#include <ctime>
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void new_world(hittable_list *world) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    new (world) hittable_list();
    world->objects = new hittable_ptr[2];
    world->add(new sphere(point3(0.0f, 0.0f, -1.0f), 0.5f));
    world->add(new sphere(point3(0.0f, -100.5f, -1.0f), 100.0f));
  }
}

__global__ void delete_world(hittable_list *world) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    for (size_t i = 0; i < world->objects_size; ++i) {
      delete world->objects[i];
    }
  }
}

__global__ void new_rand_state(unsigned long long seed, camera *cam,
                               hiprandState *rand_state) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if ((i >= cam->image_width) || (j >= cam->image_height)) {
    return;
  }

  int pixel_index = j * cam->image_width + i;
  // Each thread gets same seed, a different sequence number, no offset
  hiprand_init(seed, pixel_index, 0, &rand_state[pixel_index]);
}

int main() {

  // World
  hittable_list *world;
  checkCudaErrors(hipMalloc(&world, sizeof(*world)));
  new_world<<<1, 1>>>(world);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  // Camera
  camera *cam;
  checkCudaErrors(hipMallocManaged(&cam, sizeof(*cam)));
  new_camera<<<1, 1>>>(cam);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  const int n_thread_x = 16;
  const int n_thread_y = 16;

  std::clog << "Image Size = " << cam->image_width << "x" << cam->image_height
            << "\n";
  std::clog << "Samples Per Pixel = " << cam->samples_per_pixel << "\n";
  std::clog << "Block Dim (a x b threads) = " << n_thread_x << "x" << n_thread_y
            << "\n";

  // Frame Buffer
  color *frame_buffer;
  int n_pixels = cam->image_width * cam->image_height;
  checkCudaErrors(
      hipMallocManaged(&frame_buffer, n_pixels * sizeof(*frame_buffer)));

  // Choose Block Size and Thread Size
  int n_block_x = (cam->image_width + n_thread_x - 1) / n_thread_x;
  int n_block_y = (cam->image_height + n_thread_y - 1) / n_thread_y;
  dim3 blocks(n_block_x, n_block_y);
  dim3 threads(n_thread_x, n_thread_y);

  // Random State
  hiprandState *rand_state;
  checkCudaErrors(hipMalloc(&rand_state, n_pixels * sizeof(*rand_state)));
  new_rand_state<<<blocks, threads>>>(time(nullptr), cam, rand_state);

  // Render
  auto start = std::chrono::high_resolution_clock::now();

  render<<<blocks, threads>>>(frame_buffer, cam, world, rand_state);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  auto end = std::chrono::high_resolution_clock::now();
  auto timer_in_ms = std::chrono::duration<float, std::milli>(end - start);
  std::clog << "Time Cost = " << static_cast<int>(timer_in_ms.count() + 0.999f)
            << " ms\n";

  // Output Image
  output_image(cam, frame_buffer);

  // Cleanup Random State
  checkCudaErrors(hipFree(rand_state));

  // Cleanup Frame Buffer
  checkCudaErrors(hipFree(frame_buffer));

  // Cleanup Camera
  checkCudaErrors(hipFree(cam));

  // Cleanup World
  delete_world<<<1, 1>>>(world);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipFree(world));
}