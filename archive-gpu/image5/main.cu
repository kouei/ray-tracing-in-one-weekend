#include "hip/hip_runtime.h"
#include "color.h"
#include "cuda_utility.h"
#include "hittable.h"
#include "hittable_list.h"
#include "ray.h"
#include "rtweekend.h"
#include "sphere.h"
#include "vec3.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>

__device__ color ray_color(const ray &r, const hittable &world) {
  hit_record rec;
  if (world.hit(r, 0.0f, infinity, rec)) {
    return 0.5f * (rec.normal + color(1.0f, 1.0f, 1.0f));
  }

  vec3 unit_direction = unit_vector(r.direction());
  float a = 0.5f * (unit_direction.y() + 1.0f);
  return (1.0f - a) * color(1.0f, 1.0f, 1.0f) + a * color(0.5f, 0.7f, 1.0f);
}

__global__ void render(vec3 *frame_buffer, int image_width, int image_height,
                       vec3 pixel_delta_u, vec3 pixel_delta_v,
                       point3 pixel00_loc, point3 camera_center,
                       hittable_list *world) {

  int image_x = threadIdx.x + blockIdx.x * blockDim.x;
  int image_y = threadIdx.y + blockIdx.y * blockDim.y;
  if (image_x >= image_width || image_y >= image_height) {
    return;
  }

  int pixel_index = image_y * image_width + image_x;

  auto pixel_center =
      pixel00_loc + (image_x * pixel_delta_u) + (image_y * pixel_delta_v);
  auto ray_direction = pixel_center - camera_center;
  ray r(camera_center, ray_direction);

  color pixel_color = ray_color(r, world[0]);
  frame_buffer[pixel_index] = pixel_color;
}

__global__ void create_world(hittable_list *world) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    new (world) hittable_list();
    world->objects = new hittable_ptr[2];
    world->add(new sphere(point3(0.0f, 0.0f, -1.0f), 0.5f));
    world->add(new sphere(point3(0.0f, -100.5f, -1.0f), 100.0f));
  }
}

hittable_list *create_world_cpu() {
  hittable_list *world;
  checkCudaErrors(hipMalloc(&world, sizeof(*world)));
  create_world<<<1, 1>>>(world);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  return world;
}

int main() {

  // Image
  float aspect_ratio = 16.0f / 9.0f;
  int image_width = 1920;

  // Calculate the image height, and ensure that it's at least 1.
  int image_height = static_cast<int>(image_width / aspect_ratio);
  image_height = (image_height < 1) ? 1 : image_height;

  const int samples_per_pixel = 500;
  const int n_thread_x = 16;
  const int n_thread_y = 16;

  std::clog << "Image Size = " << image_width << "x" << image_height << "\n";
  std::clog << "Samples Per Pixel = " << samples_per_pixel << "\n";
  std::clog << "Block Dim (a x b threads) = " << n_thread_x << "x" << n_thread_y
            << "\n";

  // World

  hittable_list *world = create_world_cpu();

  // Camera

  float focal_length = 1.0f;
  float viewport_height = 2.0f;
  float viewport_width =
      viewport_height * ((float)(image_width) / image_height);
  point3 camera_center = point3(0.0f, 0.0f, 0.0f);

  // Calculate the vectors across the horizontal and down the vertical viewport
  // edges.
  vec3 viewport_u = vec3(viewport_width, 0.0f, 0.0f);
  vec3 viewport_v = vec3(0.0f, -viewport_height, 0.0f);

  // Calculate the horizontal and vertical delta vectors from pixel to pixel.
  vec3 pixel_delta_u = viewport_u / (float)(image_width);
  vec3 pixel_delta_v = viewport_v / (float)(image_height);

  // Calculate the location of the upper left pixel.
  point3 viewport_upper_left = camera_center - vec3(0.0f, 0.0f, focal_length) -
                               viewport_u / 2.0f - viewport_v / 2.0f;

  point3 pixel00_loc =
      viewport_upper_left + 0.5f * (pixel_delta_u + pixel_delta_v);

  // Allocate Frame Buffer
  int n_pixels = image_width * image_height;
  vec3 *frame_buffer;
  size_t frame_buffer_size = n_pixels * sizeof(*frame_buffer);

  checkCudaErrors(hipMallocManaged(&frame_buffer, frame_buffer_size));

  // Render

  int n_block_x = (image_width + n_thread_x - 1) / n_thread_x;
  int n_block_y = (image_height + n_thread_y - 1) / n_thread_y;
  dim3 blocks(n_block_x, n_block_y);
  dim3 threads(n_thread_x, n_thread_y);

  auto start = std::chrono::high_resolution_clock::now();

  render<<<blocks, threads>>>(frame_buffer, image_width, image_height,
                              pixel_delta_u, pixel_delta_v, pixel00_loc,
                              camera_center, world);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  auto end = std::chrono::high_resolution_clock::now();
  auto timer_in_ms = std::chrono::duration<float, std::milli>(end - start);
  std::clog << "Time Cost = " << static_cast<int>(timer_in_ms.count() + 0.999f)
            << " ms\n";

  // Output Image

  std::cout << "P3\n" << image_width << ' ' << image_height << "\n255\n";

  for (int image_y = 0; image_y < image_height; ++image_y) {
    for (int image_x = 0; image_x < image_width; ++image_x) {
      int pixel_index = image_y * image_width + image_x;
      vec3 pixel = frame_buffer[pixel_index];

      int ir = static_cast<int>(255.999 * pixel.x());
      int ig = static_cast<int>(255.999 * pixel.y());
      int ib = static_cast<int>(255.999 * pixel.z());

      std::cout << ir << ' ' << ig << ' ' << ib << '\n';
    }
  }
}