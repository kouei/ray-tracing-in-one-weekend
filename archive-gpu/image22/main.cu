#include "hip/hip_runtime.h"
#include "camera.h"
#include "color.h"
#include "cuda_utility.h"
#include "hittable.h"
#include "hittable_list.h"
#include "interval.h"
#include "material.h"
#include "material_list.h"
#include "ray.h"
#include "rtweekend.h"
#include "sphere.h"
#include "vec3.h"
#include <chrono>
#include <ctime>
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void new_world(hittable_list *world, material_list *materials) {
  if (threadIdx.x != 0 || blockIdx.x != 0) {
    return;
  }

  new (materials) material_list();
  material *material_ground = new lambertian(color(0.8f, 0.8f, 0.0f));
  material *material_center = new lambertian(color(0.1f, 0.2f, 0.5f));
  material *material_left = new dielectric(1.5f);
  material *material_right = new metal(color(0.8f, 0.6f, 0.2f), 0.0f);
  materials->add(material_ground);
  materials->add(material_center);
  materials->add(material_left);
  materials->add(material_right);

  new (world) hittable_list();
  world->add(new sphere(point3(0.0f, -100.5f, -1.0f), 100.0f, material_ground));
  world->add(new sphere(point3(0.0f, 0.0f, -1.0f), 0.5f, material_center));
  world->add(new sphere(point3(-1.0f, 0.0f, -1.0f), 0.5f, material_left));
  world->add(new sphere(point3(-1.0f, 0.0f, -1.0f), -0.4f, material_left));
  world->add(new sphere(point3(1.0f, 0.0f, -1.0f), 0.5f, material_right));
}

__global__ void delete_world(hittable_list *world, material_list *materials) {
  if (threadIdx.x != 0 || blockIdx.x != 0) {
    return;
  }

  world->~hittable_list();
  materials->~material_list();
}

__global__ void new_rand_state(unsigned long long seed, camera *cam,
                               hiprandState *rand_states) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if ((i >= cam->image_width) || (j >= cam->image_height)) {
    return;
  }

  int pixel_index = j * cam->image_width + i;
  // Each thread gets same seed, a different sequence number, no offset
  hiprand_init(seed, pixel_index, 0, &rand_states[pixel_index]);
}

int main() {

  // World
  hittable_list *world;
  checkCudaErrors(hipMalloc(&world, sizeof(*world)));
  material_list *materials;
  checkCudaErrors(hipMalloc(&materials, sizeof(*materials)));
  new_world<<<1, 1>>>(world, materials);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  // Camera
  camera *cam;
  checkCudaErrors(hipMallocManaged(&cam, sizeof(*cam)));
  new_camera<<<1, 1>>>(cam);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  // Frame Buffer
  color *frame_buffer;
  int n_pixels = cam->image_width * cam->image_height;
  checkCudaErrors(
      hipMallocManaged(&frame_buffer, n_pixels * sizeof(*frame_buffer)));

  // Choose Block Size and Thread Size
  const int n_thread_x = 16;
  const int n_thread_y = 16;
  int n_block_x = (cam->image_width + n_thread_x - 1) / n_thread_x;
  int n_block_y = (cam->image_height + n_thread_y - 1) / n_thread_y;
  dim3 blocks(n_block_x, n_block_y);
  dim3 threads(n_thread_x, n_thread_y);

  // Random State
  hiprandState *rand_states;
  checkCudaErrors(hipMalloc(&rand_states, n_pixels * sizeof(*rand_states)));
  new_rand_state<<<blocks, threads>>>(time(nullptr), cam, rand_states);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  // Render
  std::clog << "Image Size = " << cam->image_width << " x " << cam->image_height
            << "\n";
  std::clog << "Samples Per Pixel = " << cam->samples_per_pixel << "\n";
  std::clog << "Block Dim (a x b threads) = " << n_thread_x << " x "
            << n_thread_y << "\n";

  auto start = std::chrono::high_resolution_clock::now();

  render<<<blocks, threads>>>(frame_buffer, cam, world, rand_states);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  auto end = std::chrono::high_resolution_clock::now();
  auto timer_in_ms = std::chrono::duration<float, std::milli>(end - start);
  std::clog << "Time Cost = " << static_cast<int>(timer_in_ms.count() + 0.999f)
            << " ms\n";

  // Output Image
  output_image(cam, frame_buffer);

  // Cleanup Random State
  checkCudaErrors(hipFree(rand_states));

  // Cleanup Frame Buffer
  checkCudaErrors(hipFree(frame_buffer));

  // Cleanup Camera
  checkCudaErrors(hipFree(cam));

  // Cleanup World
  delete_world<<<1, 1>>>(world, materials);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipFree(world));
  checkCudaErrors(hipFree(materials));
}