#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include "vec3.h"
#include "cuda_utility.h"

__global__ void render(vec3 * frame_buffer, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) {
        return;
    }

    int pixel_index = j * max_x + i;
    frame_buffer[pixel_index] = vec3( float(i) / max_x, float(j) / max_y, 0.2f);
}

int main() {
    int nx = 1200;
    int ny = 600;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny;
    vec3 * frame_buffer;
    size_t frame_buffer_size = num_pixels * sizeof(*frame_buffer);

    // Allocate Frame Buffer
    checkCudaErrors(hipMallocManaged(&frame_buffer, frame_buffer_size));

    auto start = std::chrono::high_resolution_clock::now();
    
    // Render our buffer
    int block_dim_x = (nx + tx - 1)/ tx;
    int block_dim_y = (ny + ty - 1) / ty;
    dim3 blocks(block_dim_x, block_dim_y);
    dim3 threads(tx, ty);

    render<<<blocks, threads>>>(frame_buffer, nx, ny);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    auto end = std::chrono::high_resolution_clock::now();

    auto timer_in_ms = std::chrono::duration<float, std::milli>(end - start);
    std::cerr << "Time cost: " << timer_in_ms.count() << "ms.\n";

    // Output FrameBuffer as Image
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * frame_buffer[pixel_index].r());
            int ig = int(255.99 * frame_buffer[pixel_index].g());
            int ib = int(255.99 * frame_buffer[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    checkCudaErrors(hipFree(frame_buffer));
}