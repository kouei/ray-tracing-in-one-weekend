#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <limits>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hittable_list.h"
#include "cuda_utility.h"

constexpr const float TMAX = std::numeric_limits<float>::max();

__device__ vec3 ray_color(const ray & r, hittable_ptr_t world) {
    hit_record rec;
    if (world->hit(r, 0.0f, TMAX, rec)) {
        return 0.5f * (rec.normal + vec3(1.0f, 1.0f, 1.0f));
    }
    else {
        vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5f * (unit_direction.y() + 1.0f);
        return (1.0f - t) * vec3(1.0f, 1.0f, 1.0f) + t * vec3(0.5f, 0.7f, 1.0f);
    }
}

__global__ void render(vec3 * frame_buffer, int max_x, int max_y,
                       vec3 lower_left_corner, vec3 horizontal, vec3 vertical, vec3 origin,
                       hittable_ptr_t * world) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) {
        return;
    }

    int pixel_index = j * max_x + i;
    float u = float(i) / float(max_x);
    float v = float(j) / float(max_y);
    ray r(origin, lower_left_corner + u * horizontal + v * vertical);
    frame_buffer[pixel_index] = ray_color(r, world[0]);
}

__global__ void create_world(hittable_ptr_t * d_list, hittable_ptr_t * d_world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_list[0] = new sphere(vec3(0.0f, 0.0f, -1.0f), 0.5f);
        d_list[1] = new sphere(vec3(0.0f, -100.5f, -1.0f), 100.0f);
        d_world[0] = new hittable_list(d_list, 2);
    }
}

__global__ void free_world(hittable_ptr_t * d_list, hittable_ptr_t * d_world) {
    delete d_list[0];
    delete d_list[1];
    delete d_world[0];
}

int main() {
    int nx = 1200;
    int ny = 600;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny;
    vec3 * frame_buffer;
    size_t frame_buffer_size = num_pixels * sizeof(*frame_buffer);

    // allocate Frame Buffer
    checkCudaErrors(hipMallocManaged(&frame_buffer, frame_buffer_size));

    // make our world of hitables
    hittable_ptr_t * d_list;
    checkCudaErrors(hipMalloc(&d_list, 2 * sizeof(*d_list)));
    hittable_ptr_t * d_world;
    checkCudaErrors(hipMalloc(&d_world, sizeof(*d_world)));
    create_world<<<1, 1>>>(d_list, d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    auto start = std::chrono::high_resolution_clock::now();

    // Render our buffer
    int block_dim_x = (nx + tx - 1) / tx;
    int block_dim_y = (ny + ty - 1) / ty;
    dim3 blocks(block_dim_x, block_dim_y);
    dim3 threads(tx, ty);

    render<<<blocks, threads>>>(frame_buffer, nx, ny,
                                vec3(-2.0, -1.0, -1.0),
                                vec3(4.0, 0.0, 0.0),
                                vec3(0.0, 2.0, 0.0),
                                vec3(0.0, 0.0, 0.0),
                                d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());


    auto end = std::chrono::high_resolution_clock::now();

    auto timer_in_ms = std::chrono::duration<double, std::milli>(end - start);
    std::cerr << "Time cost: " << timer_in_ms.count() << "ms.\n";

    // Output FB as Image
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * frame_buffer[pixel_index].r());
            int ig = int(255.99 * frame_buffer[pixel_index].g());
            int ib = int(255.99 * frame_buffer[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1, 1>>>(d_list,d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(frame_buffer));

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();
}